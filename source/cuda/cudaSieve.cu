#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <sys/types.h>
#include <unistd.h>
#include <signal.h>
#include <assert.h>
#include <ctype.h>
#include <sys/time.h>

//number of threads PER BLOCK
#define NTHREADS 1024
#define NBLOCKS 1024
#define TOTALTHREADS (NTHREADS*NBLOCKS)
// uncomment the below line to see output of prime numbers
#define DEBUG

void usage();
__global__ void Sieve(char* arr, unsigned long long max, unsigned long long start);
void printResult(char* arr, unsigned long long max);

int main(int argc, char** argv){
    unsigned long long max;
    int ret;
    char* array;
    char* cuda_array;

    //makes sure user has correct input
    if (argc != 2){
        usage();
    }
    max = atoll(argv[1]);
    //creates array of size user input on CPU
    array = (char*)malloc(sizeof(char) * max);
    if(array == NULL){
        printf("malloc failed\n");
        exit(1);
    }
    //creates array of size user input on gpu
    ret = hipMalloc(&cuda_array, sizeof(char) * max);
    if (ret != hipSuccess){
      printf("hipMalloc of size %lld failed to return %d\n", max, ret);
      exit(1);
    }
    //memset all values to 1 which is the signature of being prime
    memset(array, 1, max);

    // 0 and 1 are not prime numbers
    array[0] = 0;
    array[1] = 0;

    //copy contents of CPU array into GPU array
    hipMemcpy((void*)cuda_array, array, max, hipMemcpyHostToDevice);
    unsigned long long sqrtMax;
    sqrtMax = (unsigned long long)(sqrt((double)max));
    /*can only summon so many threads at one so the for loop allows us to
     do so repetitevly*/
    for (unsigned long long i=0; i*TOTALTHREADS < sqrtMax; i++)
    {
      Sieve<<<NBLOCKS, NTHREADS>>>((char *)cuda_array, max, i*TOTALTHREADS);
    }


    //copy gpu array data to cpu array
    hipMemcpy(array, (void*)cuda_array, max, hipMemcpyDeviceToHost);

    #ifdef DEBUG
    printResult((char*)array, max);
    #endif
    //free gpu array and cpu array
    hipFree((void*)cuda_array);
    free(array);

  }
//prints out correct usage if user does not provide proper input
void usage(){
    printf("usage: ./cudaSieve [maxInt]\n");
    exit(1);
}
__global__
void Sieve(char* arr, unsigned long long max, unsigned long long start){
    unsigned long long base = blockIdx.x * blockDim.x + threadIdx.x + start;
    unsigned long long next;
    unsigned long long sqrtMax;
    sqrtMax = (unsigned long long)sqrt((double)max);
    if (base > sqrtMax){
      return;
    }
    //check if base has been marked yet, if not then mark  off all muliples of base
    if(arr[base] == 1){
      for(next = base + base; next < max; next += base){
        arr[next] = 0;
      }
    }

}
//prints out result of numbers in interval that are prime
void printResult(char* arr, unsigned long long max){
    unsigned long long i;
    for (i = 0; i < max; i++){
        if(arr[i] == 1){
            printf("%lld\n", i);
        }
    }
    printf("\n");
}
